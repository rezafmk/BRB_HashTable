#include "hip/hip_runtime.h"
#include "hashGlobal.h"

void hashtableInit(unsigned numBuckets, multipassConfig_t* mbk, unsigned groupSize)
{
	mbk->numBuckets = numBuckets;
	mbk->groupSize = groupSize;
	int numGroups = (numBuckets + (groupSize - 1)) / groupSize;
	hipMalloc((void**) &(mbk->groups), numGroups * sizeof(bucketGroup_t));
	hipMalloc((void**) &(mbk->buckets), numBuckets * sizeof(hashBucket_t*));
	hipMalloc((void**) &(mbk->dbuckets), numBuckets * sizeof(hashBucket_t*));
	hipMalloc((void**) &(mbk->locks), numBuckets * sizeof(unsigned));
	hipMalloc((void**) &(mbk->isNextDeads), numBuckets * sizeof(short));
	
	hipMemset(mbk->groups, 0, numGroups * sizeof(bucketGroup_t));
	hipMemset(mbk->dbuckets, 0, numBuckets * sizeof(hashBucket_t*));
	hipMemset(mbk->buckets, 0, numBuckets * sizeof(hashBucket_t*));
	hipMemset(mbk->locks, 0, numBuckets * sizeof(unsigned));
	hipMemset(mbk->isNextDeads, 0, numBuckets * sizeof(short));
}


__device__ unsigned int hashFunc(char* str, int len, unsigned numBuckets)
{

	int numOuterLoopIterations = len / 16;
	if(len % 16 != 0)
		numOuterLoopIterations ++;
	
	unsigned finalValue = 0;

	for(int j = 0; j < numOuterLoopIterations; j ++)
	{
		unsigned hashValue = 0;
		int startLen = 16 * j;
		int endLen = startLen + 16;
		endLen = (endLen < len)? endLen : len;

		char temp[4];
		temp[0] = (char) 0;
		temp[1] = (char) 0;
		temp[2] = (char) 0;
		temp[3] = (char) 0;

		for(int i = startLen; i < endLen; i ++)
		{
			int charCounter = (i % 16) / 4;

			if(charCounter == 0)
				charCounter = 3;
			else if(charCounter == 1)
				charCounter = 2;
			else if(charCounter == 2)
				charCounter = 1;
			else if(charCounter == 3)
				charCounter = 0;

			if(i % 4 == 0)
			{
				if(str[i] == 'C')
					temp[charCounter] = temp[charCounter] | (1 << 6);
				else if(str[i] == 'G')
					temp[charCounter] = temp[charCounter] | (1 << 7);
				else if(str[i] == 'T')
				{
					temp[charCounter] = temp[charCounter] | (1 << 7);
					temp[charCounter] = temp[charCounter] | (1 << 6);
				}

			}
			else if(i % 4 == 1)
			{
				if(str[i] == 'C')
					temp[charCounter] = temp[charCounter] | (1 << 4);
				else if(str[i] == 'G')
					temp[charCounter] = temp[charCounter] | (1 << 5);
				else if(str[i] == 'T')
				{
					temp[charCounter] = temp[charCounter] | (1 << 5);
					temp[charCounter] = temp[charCounter] | (1 << 4);
				}

			}
			else if(i % 4 == 2)
			{
				if(str[i] == 'C')
					temp[charCounter] = temp[charCounter] | (1 << 2);
				else if(str[i] == 'G')
					temp[charCounter] = temp[charCounter] | (1 << 3);
				else if(str[i] == 'T')
				{
					temp[charCounter] = temp[charCounter] | (1 << 3);
					temp[charCounter] = temp[charCounter] | (1 << 2);
				}


			}
			else
			{
				if(str[i] == 'C')
					temp[charCounter] = temp[charCounter] | (1 << 0);
				else if(str[i] == 'G')
					temp[charCounter] = temp[charCounter] | (1 << 1);
				else if(str[i] == 'T')
				{
					temp[charCounter] = temp[charCounter] | (1 << 1);
					temp[charCounter] = temp[charCounter] | (1 << 0);
				}

			}
		}

		hashValue = *((unsigned int*) &temp[0]);
		finalValue += hashValue;
	}

        return finalValue % numBuckets;
}


__device__ bool resolveSameKeyAddition(void const* key, void* value, void* oldValue, bucketGroup_t* group, multipassConfig_t* mbk)
{
	value_t* newValue = (value_t*) multipassMallocValue(sizeof(value_t), group, mbk);
	if(newValue != NULL)
	{
		newValue->documentId = ((value_t*) value)->documentId;
		newValue->next = ((value_t*) oldValue)->next;
		((value_t*) oldValue)->next = newValue;
		return true;
	}
	return false;
}

__device__ hashBucket_t* containsKey(hashBucket_t* bucket, void* key, int keySize, multipassConfig_t* mbk)
{
	while(bucket != NULL)
	{
		char* oldKey = (char*) ((largeInt) bucket + sizeof(hashBucket_t));
		bool success = true;
		//OPTIMIZE: do the comparisons 8-byte by 8-byte
		for(int i = 0; i < keySize; i ++)
		{
			if(oldKey[i] != ((char*) key)[i])
			{
				success = false;
				break;
			}
		}
		if(success)
			break;

		if(bucket->isNextDead == 0 && bucket->dnext != NULL)
			bucket = bucket->dnext;
		else
			bucket = NULL;
	}

	return bucket;
}

__device__ bool atomicAttemptIncRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	bool success;
	do
	{
		success = false;
		assume = oldRefCount;
		if(oldRefCount >= 0)
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, oldRefCount + 1);
			success = true;
		}
	} while(oldRefCount != assume);

	return success;
}

__device__ int atomicDecRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	do
	{
		assume = oldRefCount;
		if(oldRefCount >= 0) // During normal times
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, (unsigned) (oldRefCount - 1));
		}
		else // During failure
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, (unsigned) (oldRefCount + 1));
		}

	} while(oldRefCount != assume);

	return oldRefCount;
}

__device__ bool atomicNegateRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	do
	{
		assume = oldRefCount;
		if(oldRefCount >= 0)
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, ((oldRefCount * (-1)) - 1));

	} while(oldRefCount != assume);

	return (oldRefCount >= 0);
	
}

__device__ bool addToHashtable(void* key, int keySize, void* value, int valueSize, multipassConfig_t* mbk, int passno)
{
	bool success = true;
	unsigned hashValue = hashFunc((char*) key, keySize, mbk->numBuckets);

	unsigned groupNo = hashValue / mbk->groupSize;
	//unsigned groupNo = hashValue / GROUP_SIZE;

	bucketGroup_t* group = &(mbk->groups[groupNo]);
	
	hashBucket_t* existingBucket;

	int keySizeAligned = (keySize % ALIGNMET == 0)? keySize : keySize + (ALIGNMET - (keySize % ALIGNMET));
	int valueSizeAligned = (valueSize % ALIGNMET == 0)? valueSize : valueSize + (ALIGNMET - (valueSize % ALIGNMET));

	unsigned oldLock = 1;

	do
	{
		oldLock = atomicExch((unsigned*) &(mbk->locks[hashValue]), 1);

		if(oldLock == 0)
		{
			hashBucket_t* dbucket = mbk->dbuckets[hashValue];
			hashBucket_t* hbucket = mbk->buckets[hashValue];

			//First see if the key already exists in one of the entries of this bucket
			//The returned bucket is the 'entry' in which the key exists
			if(mbk->isNextDeads[hashValue] != 1 && (existingBucket = containsKey(dbucket, key, keySize, mbk)) != NULL)
			{
				void* oldValue = (void*) ((largeInt) existingBucket + sizeof(hashBucket_t) + keySizeAligned);
				if(!resolveSameKeyAddition(key, value, oldValue, group, mbk))
				{
					group->needed = 1;
					page_t* temp = group->parentPage;
					while(temp != NULL)
					{
						temp->needed = 1;
						temp = temp->next;
					}
					success = false;
				}
			}
			else
			{
				hashBucket_t* newBucket = (hashBucket_t*) multipassMalloc(sizeof(hashBucket_t) + keySizeAligned + valueSizeAligned, group, mbk);
				if(newBucket != NULL)
				{
					//TODO reduce the base offset if not null
					//newBucket->next = (bucket == NULL)? NULL : (hashBucket_t*) ((largeInt) bucket - (largeInt) mbk->dbuffer);
					newBucket->dnext = NULL;
					newBucket->next = NULL;
					if(dbucket != NULL)
					{
						newBucket->dnext = dbucket;
						newBucket->next = hbucket;
					}

					if(mbk->isNextDeads[hashValue] == 1)
						newBucket->isNextDead = 1;
					newBucket->keySize = (short) keySize;
					newBucket->valueSize = (short) valueSize;

					mbk->dbuckets[hashValue] = newBucket;
					mbk->buckets[hashValue] = (hashBucket_t*) ((largeInt) newBucket - (largeInt) mbk->dbuffer + group->parentPage->hashTableOffset);

					mbk->isNextDeads[hashValue] = 0;

					//TODO: this assumes that input key is aligned by ALIGNMENT, which is not a safe assumption
					for(int i = 0; i < (keySizeAligned / ALIGNMET); i ++)
						*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) key + i * ALIGNMET));
					for(int i = 0; i < (valueSizeAligned / ALIGNMET); i ++)
						*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned + i * ALIGNMET)) = *((largeInt*) ((largeInt) value + i * ALIGNMET));
				}
				else
				{
					success = false;
				}
			}

			atomicExch((unsigned*) &(mbk->locks[hashValue]), 0);
		}
	} while(oldLock == 1);

	return success;
}

__global__ void setGroupsPointersDead(multipassConfig_t* mbk, unsigned numBuckets)
{
	int index = TID;
	if(index < numBuckets)
	{
		int groupNo = index / mbk->groupSize;
		if(mbk->groups[groupNo].needed == 0)
			mbk->isNextDeads[index] = 1;
	}
	
}



multipassConfig_t* initMultipassBookkeeping(int* hostCompleteFlag, 
						int* gpuFlags, 
						int flagSize,
						int numThreads,
						int epochNum,
						int numRecords,
						int pagePerGroup)
{
	
	multipassConfig_t* mbk = (multipassConfig_t*) malloc(sizeof(multipassConfig_t));
	mbk->hostCompleteFlag = hostCompleteFlag;
	mbk->gpuFlags = gpuFlags;
	mbk->flagSize = flagSize;
	mbk->numThreads = numThreads;
	mbk->epochNum = epochNum;
	mbk->numRecords = numRecords;


	mbk->availableGPUMemory = (230 * (1 << 20));
	mbk->hhashTableBufferSize = MAX_NO_PASSES * mbk->availableGPUMemory;
	mbk->hhashTableBaseAddr = malloc(mbk->hhashTableBufferSize);
	memset(mbk->hhashTableBaseAddr, 0, mbk->hhashTableBufferSize);
	mbk->hashTableOffset = (largeInt) mbk->hhashTableBaseAddr;

	//This is how we decide the number of groups: based on the number of available pages, we make sure 
	//group size is calculated in a way that a given number of `pagePerGroup` pages are assigned to each group
	int availableNumPages = mbk->availableGPUMemory / PAGE_SIZE;
	mbk->groupSize = (pagePerGroup * NUM_BUCKETS) / availableNumPages;
	mbk->numGroups = (NUM_BUCKETS + (mbk->groupSize - 1)) / mbk->groupSize;
	//mbk->numGroups = (NUM_BUCKETS + (GROUP_SIZE - 1)) / GROUP_SIZE;
	printf("############# groupSize: %d, number of groups: %d\n", mbk->groupSize, mbk->numGroups);


	hipMalloc((void**) &(mbk->dfailedFlag), sizeof(bool));
	hipMemset(mbk->dfailedFlag, 0, sizeof(bool));


	hipMalloc((void**) &(mbk->depochSuccessStatus), epochNum * sizeof(char));
	hipMemset(mbk->depochSuccessStatus, 0, epochNum * sizeof(char));
	mbk->epochSuccessStatus = (char*) malloc(epochNum * sizeof(char));


	// Calling initPaging
	initPaging(mbk->availableGPUMemory, mbk);

	hashtableInit(NUM_BUCKETS, mbk, mbk->groupSize);
	
	
	printf("@INFO: transferring config structs to GPU memory\n");

	hipMalloc((void**) &(mbk->dstates), mbk->numRecords * sizeof(char));
	hipMemset(mbk->dstates, 0, mbk->numRecords * sizeof(char));


	mbk->myNumbers = (int*) malloc(2 * numThreads * sizeof(int));
	hipMalloc((void**) &(mbk->dmyNumbers), 2 * numThreads * sizeof(int));
	hipMemset((mbk->dmyNumbers), 0, 2 * numThreads * sizeof(int));

	size_t total, free;
	hipMemGetInfo(&free, &total);
	printf("total memory: %luMB, free: %luMB\n", total / (1 << 20), free / (1 << 20));


	printf("@INFO: number of page: %d\n", (int)(mbk->availableGPUMemory / PAGE_SIZE));
	printf("@INFO: number of hash groups: %d\n", mbk->numGroups);

	return mbk;
}

bool checkAndResetPass(multipassConfig_t* mbk, multipassConfig_t* dmbk)
{
	hipError_t errR = hipGetLastError();
	printf("#######Error at the beginning of checkAndReset: %s\n", hipGetErrorString(errR));

	hipMemcpy(mbk, dmbk, sizeof(multipassConfig_t), hipMemcpyDeviceToHost);
	bool failedFlag = false;
	int* hostCompleteFlag = mbk->hostCompleteFlag;
	int* gpuFlags = mbk->gpuFlags;
	bool* dfailedFlag = mbk->dfailedFlag;
	int* dmyNumbers = mbk->dmyNumbers;
	int* myNumbers = mbk->myNumbers;
	int flagSize = mbk->flagSize;
	void* hhashTableBaseAddr = mbk->hhashTableBaseAddr;
	largeInt hhashTableBufferSize = mbk->hhashTableBufferSize;
	int numThreads = mbk->numThreads;
	char* epochSuccessStatus = mbk->epochSuccessStatus;
	char* depochSuccessStatus = mbk->depochSuccessStatus;
	int epochNum = mbk->epochNum;

	hipMemcpy(epochSuccessStatus, depochSuccessStatus, epochNum * sizeof(char), hipMemcpyDeviceToHost);
	for(int i = 0; i < epochNum; i ++)
	{
		if(epochSuccessStatus[i] == UNTESTED)
			epochSuccessStatus[i] = SUCCEED;
		else if(epochSuccessStatus[i] == FAILED)
			epochSuccessStatus[i] = UNTESTED;
	}
	hipMemcpy(depochSuccessStatus, epochSuccessStatus, epochNum * sizeof(char), hipMemcpyHostToDevice);


	memset((void*) hostCompleteFlag, 0, flagSize);
	hipMemset(gpuFlags, 0, flagSize / 2);

	hipMemcpy(&failedFlag, dfailedFlag, sizeof(bool), hipMemcpyDeviceToHost);
	hipMemset(dfailedFlag, 0, sizeof(bool));

	hipMemcpy(mbk->hpages, mbk->pages, mbk->totalNumPages * sizeof(page_t), hipMemcpyDeviceToHost);

	
	hipMemcpy(mbk->hfreeListId, mbk->freeListId, mbk->totalNumPages * sizeof(int), hipMemcpyDeviceToHost);

	int freeListCounter = 0;
	int neededCounter = 0;
	int unneededCounter = 0;
	for(int i = 0; i < mbk->totalNumPages; i ++)
	{
		if(mbk->hpages[i].needed == 0)
		{
			hipMemcpy((void*) ((largeInt) mbk->hpages[i].hashTableOffset + mbk->hpages[i].id * PAGE_SIZE), (void*) ((largeInt) mbk->dbuffer + mbk->hpages[i].id * PAGE_SIZE), PAGE_SIZE, hipMemcpyDeviceToHost);
			hipMemset((void*) ((largeInt) mbk->dbuffer + mbk->hpages[i].id * PAGE_SIZE), 0, PAGE_SIZE);

			mbk->hpages[i].hashTableOffset += mbk->totalNumPages * PAGE_SIZE;
			mbk->hpages[i].next = NULL;
			mbk->hpages[i].used = 0;

			mbk->hfreeListId[freeListCounter ++] = mbk->hpages[i].id;
			unneededCounter ++;
		}
		else
		{
			mbk->hpages[i].needed = 0;
			//printf("Page %d is needed..\n", i);
			neededCounter ++;
		}
	}

	printf("@INFO: number of needed pages: %d, and number of unneededpages: %d (number of groups: %d)\n", neededCounter, unneededCounter, NUM_BUCKETS / mbk->groupSize);

	hipMemcpy(mbk->freeListId, mbk->hfreeListId, mbk->totalNumPages * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mbk->pages, mbk->hpages, mbk->totalNumPages * sizeof(page_t), hipMemcpyHostToDevice);
	mbk->totalNumFreePages = freeListCounter;
	

	printf("totalnoPage * pagesize: %llu, hhashtbufferSize: %llu\n", (largeInt) mbk->totalNumPages * PAGE_SIZE, (largeInt) hhashTableBufferSize);
	mbk->hashTableOffset += mbk->totalNumPages * PAGE_SIZE;
	if((mbk->hashTableOffset + mbk->totalNumPages * PAGE_SIZE) > ((largeInt) hhashTableBaseAddr + hhashTableBufferSize) && failedFlag)
	{
		printf("Need more space on CPU memory for the hash table. Aborting...\n");
		exit(1);
	}


	mbk->initialPageAssignedCounter = 0;


	errR = hipGetLastError();
	printf("#######Error before setGroupPointer is: %s\n", hipGetErrorString(errR));

	printf("Before calling setGroupPointer, number of grids: %d\n", ((NUM_BUCKETS) + 1023) / 1024);
	setGroupsPointersDead<<<(((NUM_BUCKETS) + 1023) / 1024), 1024>>>(dmbk, NUM_BUCKETS);
	//setGroupsPointersDead<<<(((NUM_BUCKETS) + 256) / 255), 256>>>(mbk->groups, NUM_BUCKETS, GROUP_SIZE);
	hipDeviceSynchronize();

	errR = hipGetLastError();
	printf("#######Error after setGroupPointer is: %s\n", hipGetErrorString(errR));

	hipMemcpy(myNumbers, dmyNumbers, 2 * numThreads * sizeof(int), hipMemcpyDeviceToHost);
	hipMemset(dmyNumbers, 0, 2 * numThreads * sizeof(int));

	largeInt totalSuccess = 0;
	largeInt totalFailed = 0;
	for(int i = 0; i < numThreads; i ++)
	{
		totalSuccess += myNumbers[i * 2];
		totalFailed += myNumbers[i * 2 + 1];
	}

	printf("Total success: %lld\n", totalSuccess);
	printf("Total failure: %lld\n", totalFailed);

	hipMemcpy(dmbk, mbk, sizeof(multipassConfig_t), hipMemcpyHostToDevice);

	return failedFlag;
}

void* getKey(hashBucket_t* bucket)
{
	return (void*) ((largeInt) bucket + sizeof(hashBucket_t));
}

void* getValue(hashBucket_t* bucket)
{
	int keySizeAligned = (bucket->keySize % ALIGNMET == 0)? bucket->keySize : bucket->keySize + (ALIGNMET - (bucket->keySize % ALIGNMET));
	return (void*) ((largeInt) bucket + sizeof(hashBucket_t) + keySizeAligned);
}


