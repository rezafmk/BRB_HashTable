#include "hip/hip_runtime.h"
#include "global.h"

void hashtableInit(int numBuckets, hashtableConfig_t* hconfig)
{
	hconfig->numBuckets = numBuckets;
	int numGroups = (numBuckets + (GROUP_SIZE - 1)) / GROUP_SIZE;
	hipMalloc((void**) &(hconfig->groups), numGroups * sizeof(bucketGroup_t));
	hipMemset(hconfig->groups, 0, numGroups * sizeof(bucketGroup_t));
	//hconfig->groups = (bucketGroup_t*) malloc(numGroups * sizeof(bucketGroup_t));
}

__device__ unsigned int hashFunc(char* str, int len, unsigned numBuckets)
{
        unsigned hash = 2166136261;
        unsigned FNVMultiple = 16777619;

        for(int i = 0; i < len; i ++)
        {
                char c = str[i];

                hash += (int) c;
                hash = hash * FNVMultiple;  /* multiply by the magic number */
                hash += len;
                hash -= (int) c;
        }

        return hash % numBuckets;
}


__device__ void resolveSameKeyAddition(void const* key, void* value, void* oldValue)
{
	*((int*) oldValue) += 1;
	printf("oldValue: %d\n", *((int*) oldValue));
}

__device__ hashBucket_t* containsKey(hashBucket_t* bucket, void* key, int keySize)
{
	while(bucket != NULL)
	{
		char* oldKey = (char*) ((largeInt) bucket + sizeof(hashBucket_t));
		bool success = true;
		//OPTIMIZE: do the comparisons 8-byte by 8-byte
		for(int i = 0; i < keySize; i ++)
		{
			if(oldKey[i] != ((char*) key)[i])
			{
				success = false;
				break;
			}
		}
		if(success)
			break;

		bucket = bucket->next;
	}

	return bucket;
}

__device__ bool atomicAttemptIncRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	bool success;
	do
	{
		success = false;
		assume = oldRefCount;
		if(oldRefCount >= 0)
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, oldRefCount + 1);
			success = true;
		}
	} while(oldRefCount != assume);

	return success;
}

__device__ int atomicDecRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	do
	{
		assume = oldRefCount;
		if(oldRefCount >= 0) // During normal times
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, (unsigned) (oldRefCount - 1));
		}
		else // During failure
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, (unsigned) (oldRefCount + 1));
		}

	} while(oldRefCount != assume);

	return oldRefCount;
}

__device__ void atomicNegateRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	do
	{
		assume = oldRefCount;
		if(oldRefCount >= 0)
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, -oldRefCount);

	} while(oldRefCount != assume);
	
}

__device__ bool addToHashtable(void* key, int keySize, void* value, int valueSize, hashtableConfig_t* hconfig, pagingConfig_t* pconfig)
{
	bool success = true;
	unsigned hashValue = hashFunc((char*) key, keySize, hconfig->numBuckets);

	unsigned groupNo = hashValue / GROUP_SIZE;
	unsigned offsetWithinGroup = hashValue % GROUP_SIZE;

	bucketGroup_t* group = &(hconfig->groups[groupNo]);
	
	// Incrementing the `refCount`. If refCount was negative (which means group is failed), return
	if(atomicAttemptIncRefCount(&(group->refCount)) != true)
		return false;

	hashBucket_t* bucket = group->buckets[offsetWithinGroup];
	hashBucket_t* existingBucket;

	int keySizeAligned = (keySize % ALIGNMET == 0)? keySize : keySize + (ALIGNMET - (keySize % ALIGNMET));
	int valueSizeAligned = (valueSize % ALIGNMET == 0)? valueSize : valueSize + (ALIGNMET - (valueSize % ALIGNMET));

	unsigned oldLock = 1;

	do
	{
		oldLock = atomicExch((unsigned*) &(group->locks[offsetWithinGroup]), 1);

		if(oldLock == 0)
		{
			//First see if the key already exists in one of the entries of this bucket
			//The returned bucket is the 'entry' in which the key exists
			if((existingBucket = containsKey(bucket, key, keySize)) != NULL)
			{
				void* oldValue = (void*) ((largeInt) existingBucket + sizeof(hashBucket_t) + keySizeAligned);
				resolveSameKeyAddition(key, value, oldValue);
			}
			else
			{
				hashBucket_t* newBucket = (hashBucket_t*) multipassMalloc(sizeof(hashBucket_t) + keySizeAligned + valueSizeAligned, group, pconfig);

				if(newBucket != NULL)
				{
					//TODO reduce the base offset if not null
					//newBucket->next = (bucket == NULL)? NULL : (hashBucket_t*) ((largeInt) bucket - (largeInt) pconfig->dbuffer);
					//group->failed = 1;
					//revokePage(group->parentPage, pconfig); //TODO uncomment
					newBucket->next = bucket;
					group->buckets[offsetWithinGroup] = newBucket;

					//TODO: this assumes that input key is aligned by ALIGNMENT, which is not a safe assumption
					for(int i = 0; i < (keySizeAligned / ALIGNMET); i ++)
						*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) key + i * ALIGNMET));
					for(int i = 0; i < (valueSizeAligned / ALIGNMET); i ++)
						*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned + i * ALIGNMET)) = *((largeInt*) ((largeInt) value + i * ALIGNMET));
				}
				else
				{
					atomicNegateRefCount(&(group->refCount));
					success = false;
				}
			}

			atomicExch((unsigned*) &(group->locks[offsetWithinGroup]), 0);
		}
	} while(oldLock == 1);

	int oldRefCount = atomicDecRefCount(&(group->refCount));
	if(oldRefCount == -1)
	{
		long long unsigned pageAddress = (long long unsigned) group->parentPage;
		page_t* oldPage = (page_t*) atomicCAS((long long unsigned*) &(group->parentPage), pageAddress, NULL);
		revokePage(oldPage, pconfig);
	}

	return success;
}


