#include <stdio.h>
#include <unistd.h>
#include <pthread.h>
#include <time.h>
#include <sys/time.h>

typedef long long int largeInt;

#define RECORD_LENGTH 64
#define NUM_BUCKETS 1000000

typedef struct
{
	pagingConfig_t* pconfig;
	hipStream_t* serviceStream;
} dataPackage_t;

typedef struct
{
	struct hashEntry_t* next;
	int value;	
	char key[128];
} hashEntry_t;

typedef struct
{
	hashEntry_t* entry;
	unsigned lock;
} hashBucket_t;


unsigned hashFunc(char* str, int len, int numBuckets)
{
        unsigned hash = 2166136261;
        unsigned FNVMultiple = 16777619;

        for(int i = 0; i < len; i ++)
        {
                char c = str[i];

                hash += (int) c;
                hash = hash * FNVMultiple;  /* multiply by the magic number */
                hash += len;
                hash -= (int) c;
        }

        return hash % numBuckets;
}

hashBucket_t* containsKey(hashBucket_t* bucket, char* key, int keySize)
{
	while(bucket != NULL)
	{
		char* oldKey = (char*) bucket->key;
		bool success = true;
		//OPTIMIZE: do the comparisons 8-byte by 8-byte
		for(int i = 0; i < keySize; i ++)
		{
			if(oldKey[i] != key[i])
			{
				success = false;
				break;
			}
		}
		if(success)
			break;

		bucket = bucket->next;
	}

	return bucket;

}

// Allocation of hash table
	
bool addToHashtable(hashBucket_t* hashTable, char* key, int keySize, int value, int valueSize)
{
	bool success = true;
	unsigned hashValue = hashFunc((char*) key, keySize, hconfig->numBuckets);

	
	hashBucket_t* bucket = &hashTable[hashValue];
	hashEntry_t* entry = bucket->entry;


	int keySizeAligned = (keySize % ALIGNMET == 0)? keySize : keySize + (ALIGNMET - (keySize % ALIGNMET));
	int valueSizeAligned = (valueSize % ALIGNMET == 0)? valueSize : valueSize + (ALIGNMET - (valueSize % ALIGNMET));

	unsigned oldLock = 1;

	do
	{
		oldLock = __sync_lock_test_and_set(&(bucket->lock), (unsigned) 1);
		
	} while(oldLock == 1);
	
	hashEntry_t* existingEntry;
	//First see if the key already exists in one of the entries of this bucket
	//The returned bucket is the 'entry' in which the key exists
	if((existingEntry = containsKey(entry, key, keySize)) != NULL)
	{
		existingEntry->value ++;
	}
	else
	{
		hashEntry_t* newEntry = (hashEntry_t*) malloc(sizeof(hashEntry_t));

		if(newEntry != NULL)
		{
			newEntry->next = entry;
			for(int j = 0; j < keySize; j ++)
				newEntry->key[j] = key[j];

			newEntry->value = value;
			bucket->entry = newEntry;
		}
		else
		{
			printf("Failed to malloc\n");
			success = false;
		}
	}

	bucket->lock = 0;

	return success;
}


void* kernel(void* arg)//char* records, int numRecords, int* recordSizes, int numThreads, pagingConfig_t* pconfig, hashtableConfig_t* hconfig, int* status)
{
	dataPackage_t* argument = (dataPackage_t*) arg;
	char* recrods = argument->records;
	int numRecords = argument->numRecords;
	int* recordSizes  = argument->recordSizes;
	int numThreads = argument->numThreads;
	hashEntry_t* hashTable = agument->hashTable;
	int index = argument->index;
	
	
	for(int i = index; i < numRecords; i += numThreads)
	{
		char* record = &records[i * RECORD_LENGTH];
		int recordSize = recordSizes[i];
		recordSize = (recordSize % 8 == 0)? recordSize : (recordSize + (8 - (recordSize % 8)));
		largeInt value = 1;
		if(addToHashtable((void*) record, recordSize, (void*) &value, sizeof(largeInt), hconfig, pconfig) == true)
			status[index * 2] ++;
		else
			status[index * 2 + 1] ++;
	}
}


int main(int argc, char** argv)
{
	hipError_t errR;
	int numRecords = 4500000;
	if(argc == 2)
	{
		numRecords = atoi(argv[1]);
	}	

	dim3 grid(8, 1, 1);
	dim3 block(512, 1, 1);
	int numThreads = grid.x * block.x;
	numRecords = (numRecords % numThreads == 0)? numRecords : (numRecords + (numThreads - (numRecords % numThreads)));
	printf("@INFO: Number of records: %d (%d per thread)\n", numRecords, numRecords / numThreads);
	
	

	printf("@INFO: Allocating %dMB for input data\n", (numRecords * RECORD_LENGTH) / (1 << 20));
	char* records = (char*) malloc(numRecords * RECORD_LENGTH);
	int* recordSizes = (int*) malloc(numRecords * sizeof(int));

	srand(time(NULL));

	for(int i = 0; i < numRecords; i ++)
	{
		recordSizes[i] = rand() % (RECORD_LENGTH - 8);
		if(recordSizes[i] < 14)
			recordSizes[i] = 14;
	}

	for(int i = 0; i < numRecords; i ++)
	{
		records[i * RECORD_LENGTH + 0] = 'w';
		records[i * RECORD_LENGTH + 1] = 'w';
		records[i * RECORD_LENGTH + 2] = 'w';
		records[i * RECORD_LENGTH + 3] = '.';

		int j = 4;
		for(; j < recordSizes[i] - 4; j ++)
			records[i * RECORD_LENGTH + j] = rand() % 26 + 97;

		records[i * RECORD_LENGTH + j + 0] = '.';
		records[i * RECORD_LENGTH + j + 1] = 'c';
		records[i * RECORD_LENGTH + j + 2] = 'o';
		records[i * RECORD_LENGTH + j + 3] = 'm';
	}

	printf("Some records:\n");
	for(int i = 0; i < 20; i ++)
	{
		for(int j = 0; j < recordSizes[i]; j ++)
		{
			printf("%c", records[i * RECORD_LENGTH + j]);
		}
		printf("\n");
	}

	printf("@INFO: done initializing the input data\n");
	hashBucket_t* hashTable = (hashBucket_t*) malloc(NUM_BUCKETS * sizeof(hashBucket_t));
	int* status = (int*) malloc(numThreads * 2 * sizeof(int));


	//==========================================================================//
	
	 struct timeval partial_start, partial_end;//, exec_start, exec_end;
        time_t sec;
        time_t ms;
        time_t diff;
	


	//====================== Calling the kernel ================================//


	//Spawn the pthread functions

	
	pthread_t thread;
	dataPackage_t argument;

	argument.pconfig = pconfig;
	argument.serviceStream = &serviceStream;

	pthread_create(&thread, NULL, recyclePages, &argument);


	//Join threads here

	gettimeofday(&partial_end, NULL);
        sec = partial_end.tv_sec - partial_start.tv_sec;
        ms = partial_end.tv_usec - partial_start.tv_usec;
        diff = sec * 1000000 + ms;

        printf("\n%10s:\t\t%0.0f\n", "Total time", (double)((double)diff/1000.0));



	int totalSuccess = 0, totalFailed = 0;
	for(int i = 0; i < numThreads; i ++)
	{
		totalSuccess += status[i * 2];
		totalFailed += status[i * 2 + 1];
	}

	printf("Total success: %d\n", totalSuccess);
	printf("Total failed: %d\n", totalFailed);

	return 0;
}
