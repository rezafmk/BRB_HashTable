#include "hip/hip_runtime.h"
#include "global.h"
#include "hashGlobal.h"
#include "kernel.cu"

#define TEXTITEMSIZE 1
#define DATAITEMSIZE 1
#define RECORD_SIZE 64
#define NUM_BUCKETS 10000000
#define GB 1073741824

#define EPOCHCHUNK 30

#define NUMTHREADS (MAXBLOCKS * BLOCKSIZE)
#define DISPLAY_RESULTS


__global__ void wordCountKernelMultipass(
				char* data, 
				int numRecords,
				ptr_t* textAddrs,
				char* textData,
				int volatile* completeFlag,
				int* gpuFlags,
				strides_t* stridesSpace1,
				firstLastAddr_t* firstLastAddrsSpace1,
				int iterations,
				int epochNum, 
				int* myNumbers,
				int numThreads,
				pagingConfig_t* pconfig,
				hashtableConfig_t* hconfig,
				char* states,
				bool* failedFlag,
				char* epochSuccessStatus
				)
{
	int index = TID2;
	bool prediction = (threadIdx.x < BLOCKSIZE);

	int chunkSize = numRecords / numThreads;
	chunkSize = (numRecords % numThreads == 0)? chunkSize : chunkSize + 1;
	int start = index * chunkSize;
	int end = start + chunkSize;
	end = (end > numRecords)? numRecords : end;

	int genericCounter;
	
	int flagGPU[3];
	flagGPU[0] = 1;
	flagGPU[1] = 1;
	flagGPU[2] = 1;

	int flagCPU[3];
	flagCPU[0] = 1;
	flagCPU[1] = 1;
	flagCPU[2] = 1;

	__shared__ int addrDis1[PATTERNSIZE * BLOCKSIZE];
	__shared__ bool validatedArray[BLOCKSIZE / WARPSIZE];

	int s = 0;

	ptr_t previousAddrSpace1;
	ptr_t firstAddrSpace1;
	int i = start;

	for(int j = 0; i < end; j ++)
	{
#if 1
		if((prediction && j < epochNum && epochSuccessStatus[j] == (char) 1) || (!prediction && j > 1 && epochSuccessStatus[j - 2] == (char) 1))
		{
			i += iterations;
			continue;
		}
#endif

		if(prediction && j < epochNum)	
		{

			genericCounter = (blockIdx.x * BLOCKSIZE + (threadIdx.x / 32) * WARPSIZE) * iterations + (threadIdx.x % 32);
			if(threadIdx.x == 0)
				for(int m = 0; m < BLOCKSIZE / WARPSIZE; m ++)
					validatedArray[m] = true;
			__threadfence_block();

			previousAddrSpace1 = 0;
			bool firstTimeSpace1 = true;
			int addrCounterSpace1 = 0;

			int ihistory = i; 
			int loopCounter = 0;
			for(; (loopCounter < iterations) && (i < end); loopCounter ++, i ++)
			{
				ptr_t addr = (ptr_t) &data[i * RECORD_SIZE];
				if(addrCounterSpace1 < PATTERNSIZE)
					addrDis1[(threadIdx.x % BLOCKSIZE) * PATTERNSIZE + addrCounterSpace1] = (int) (addr - previousAddrSpace1);

				previousAddrSpace1 = addr;
				addrCounterSpace1 ++;
				if(firstTimeSpace1)
				{
					addrCounterSpace1 --;
					firstTimeSpace1 = false;
					firstAddrSpace1 = previousAddrSpace1;
				}
			}

			int strideSizeSpace1 = findPatternKernel(&addrDis1[(threadIdx.x % BLOCKSIZE) * PATTERNSIZE], PATTERNSIZE);
			bool validated = true;
			int strideCounterSpace1 = 0;
			previousAddrSpace1 = firstAddrSpace1;
			int dataCountSpace1 = 0;

			i = ihistory;
			loopCounter = 0;
			for(; (loopCounter < iterations) && (i < end); loopCounter ++, i ++)
			{
				ptr_t addr = (ptr_t) &data[i * RECORD_SIZE];
				dataCountSpace1 ++;
				if(addr != previousAddrSpace1)
					validated = false;
				previousAddrSpace1 += addrDis1[strideCounterSpace1 % strideSizeSpace1];
				strideCounterSpace1 ++;
			}
			(stridesSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].strideSize = strideSizeSpace1;

			if(!validated)
				validatedArray[(threadIdx.x / 32)] = false;
			__threadfence_block();
			(firstLastAddrsSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].firstAddr = firstAddrSpace1;
			(firstLastAddrsSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].lastAddr = previousAddrSpace1;
			(firstLastAddrsSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].itemCount = dataCountSpace1;

			if(validatedArray[(threadIdx.x / 32)])
			{
				bool equalPattern = true;
				for(int j = 0; j < 32; j ++)
				{
					//can be aggressively optimized!!
					for(int m = 0; m < strideSizeSpace1; m ++)
						if(addrDis1[(threadIdx.x % BLOCKSIZE) * PATTERNSIZE + m] != addrDis1[(((threadIdx.x % BLOCKSIZE) / WARPSIZE) * WARPSIZE + j) * PATTERNSIZE + m])
							equalPattern = false;
				}

				if(equalPattern)
				{
					if((threadIdx.x % 32) != 0)
					{
						for(int m = 0; m < strideSizeSpace1; m ++)
							addrDis1[(threadIdx.x % BLOCKSIZE) * PATTERNSIZE + m] = -1;
					}
				}

				for(int m = 0; m < strideSizeSpace1; m ++)
					(stridesSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].strides[m] = addrDis1[(threadIdx.x % BLOCKSIZE) * PATTERNSIZE + m];
				(stridesSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].strideSize = strideSizeSpace1;
			}
			else
			{
				for(int m = 0; m < (stridesSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].strideSize; m ++)
					(stridesSpace1 + (s * (blockDim.x / 2) * gridDim.x))[index].strides[m] = -1;

				i = ihistory;
				loopCounter = 0;
				for(; (loopCounter < iterations) && (i < end); loopCounter ++, i ++)
				{
					(textAddrs + (s * (iterations * (blockDim.x / 2) * gridDim.x)))[genericCounter] = (ptr_t) &data[i];
					genericCounter += 32;
				}
			}

		}

		if(prediction)
			asm volatile("bar.sync %0, %1;" ::"r"(4), "r"(blockDim.x / 2)); 


		if(threadIdx.x == 0 && j < epochNum)
		{
			flagGPU[s] *= -1;
			completeFlag[blockIdx.x * 12 + s * 2] = flagGPU[s];
			__threadfence_system();
		}
	

		if(prediction && j < epochNum)
			s = (s + 1) % 3;


		if(!prediction && threadIdx.x == BLOCKSIZE && j > 1)
		{
			flagCPU[s] *= -1;
			volatile int value = 0;
			do
			{
				asm volatile("ld.global.cg.u32 %0, [%1];" :"=r"(value) :"l"(&gpuFlags[blockIdx.x * 6 + s]));
				
			} while(value != flagCPU[s]);
		}

		if(!prediction)
			asm volatile("bar.sync %0, %1;" ::"r"(5), "r"(blockDim.x / 2));

		if(!prediction && j > 1)
		{
			genericCounter = ((blockIdx.x * BLOCKSIZE + ((threadIdx.x - (blockDim.x / 2)) / WARPSIZE) * WARPSIZE) * iterations) * RECORD_SIZE + (threadIdx.x % 32) * COPYSIZE;
			int step = 0;

			int loopCounter = 0;
			for(; (loopCounter < iterations) && (i < end); loopCounter ++, i ++)
			{
				//TODO: since the hash table lib is ours, we can read the data in it coalescly.
				char URL[64];
				int urlSize = 0;
				for(int k = 0; k < RECORD_SIZE; k ++)
				{
					if(states[i] == (char) 0)
					{
						char c = (textData + (s * iterations * RECORD_SIZE * (blockDim.x / 2) * gridDim.x))[genericCounter + step];
						URL[k] = c;
						if(c != ' ' && c != '\n')
							urlSize ++;
					}
					//sum += (int) c;

					step ++;
					genericCounter += (step / COALESCEITEMSIZE) * (WARPSIZE * COALESCEITEMSIZE);
					step %= COALESCEITEMSIZE;
				}

				if(states[i] == (char) 0)
				{
					largeInt value = 1;
					if(addToHashtable((void*) URL, urlSize, (void*) &value, sizeof(largeInt), hconfig, pconfig) == true)
					{
						myNumbers[index * 2] ++;
						states[i] = SUCCEED;
					}
					else
					{
						myNumbers[index * 2 + 1] ++;
						*failedFlag = true;
						epochSuccessStatus[j - 2] = FAILED;
					}
				}
					
			}
				

			s = (s + 1) % 3;
		}

		__syncthreads();
	}
}

int countToNextWord(char* start)
{
	int counter = 0;
	while(start[counter] != ' ')
		counter ++;
	while(start[counter] < 'a' || start[counter] > 'z')
		counter ++;

	return counter;
}

struct timeval global_start[MAXBLOCKS];//, global_end;


void startGlobalTimer(int tid)
{
	gettimeofday(&global_start[tid], NULL);
}

void endGlobalTimer(int tid, char* message)
{
	struct timeval end;
	time_t sec, ms, diff;

	gettimeofday(&end, NULL);
	sec = end.tv_sec - global_start[tid].tv_sec;
	ms = end.tv_usec - global_start[tid].tv_usec;
	diff = sec * 1000000 + ms;

	//printf("[%d] %10s:\t\t%0.1fms\n", tid, message, (double)((double)diff/1000.0));
	//fflush(stdout);
}

void* copyMethodPattern(void* arg)
{
	copyPackagePattern* pkg = (copyPackagePattern*) arg;

	char* fdata = pkg->srcSpace;
	int myBlock = pkg->myBlock;
	int epochDuration = pkg->epochDuration;
	//strides_t* stridesSpace[1];
	//stridesSpace[0] = pkg->stridesSpace[0];
	//long long unsigned int sourceSpaceSize1 = pkg->sourceSpaceSize1;

	firstLastAddr_t* firstLastAddrsSpace[1];
	firstLastAddrsSpace[0] = pkg->firstLastAddrsSpace[0];

	unsigned warpStart = pkg->warpStart;
	unsigned warpEnd = pkg->warpEnd;

	unsigned spaceDataItemSizes[1];
	spaceDataItemSizes[0] = pkg->spaceDataItemSizes[0];

	char* hostBuffer[1];
	hostBuffer[0] = pkg->hostBuffer[0];


	for(int k = warpStart; k < warpEnd; k ++)
	{
		//strides_t* warpStrides = &(stridesSpace[0][myBlock * BLOCKSIZE + k * WARPSIZE]);
		firstLastAddr_t* warpFirstLastAddrs = &(firstLastAddrsSpace[0][myBlock * BLOCKSIZE + k * WARPSIZE]);
		unsigned int curAddrs;
		//int strideCounter = 0;
		unsigned int offset;

		for(int i = 0; i < WARPSIZE; i ++)
		{
			curAddrs = warpFirstLastAddrs[i].firstAddr;

			//1
			offset = ((myBlock * BLOCKSIZE + k * WARPSIZE) * epochDuration) * spaceDataItemSizes[0] + i * COPYSIZE;

			copytype_t* tempSpace = (copytype_t*) &hostBuffer[0][offset];

			//TODO this has to use strides to know what address to load next.
			for(int j = 0; j < warpFirstLastAddrs[i].itemCount; j ++)
			{
				for(int m = 0; m < RECORD_SIZE / COPYSIZE; m ++)
				{
					tempSpace[(j * (RECORD_SIZE / COPYSIZE) + m) * WARPSIZE] = *((copytype_t*) &fdata[(curAddrs + j * RECORD_SIZE + m * COPYSIZE)]);
				}
			}

		}
	}


	return NULL;
}

void* pipelineData(void* argument)
{

        dataPackage* threadData = (dataPackage*) argument;
	//endGlobalTimer(threadData->myBlock, "@@ Thread creation");

        hipStream_t* streamPtr = threadData->streamPtr;
        int volatile * volatile flags = threadData->flags;
	int* gpuFlags = threadData->gpuFlags;
	hipStream_t* execStream = threadData->execStream;

	char* textData[3];
	textData[0] = threadData->textData[0];
	textData[1] = threadData->textData[1];
	textData[2] = threadData->textData[2];

	char* gpuSpaces[1][3];
	gpuSpaces[0][0] = textData[0];
	gpuSpaces[0][1] = textData[1];
	gpuSpaces[0][2] = textData[2];

        char* fdata = threadData->fdata;
        int myBlock = threadData->myBlock;

        int threadBlockSize = threadData->threadBlockSize;

	unsigned int epochDuration = threadData->epochDuration;

	int textItemSize = threadData->textItemSize;

	char* textHostBuffer[3];
	textHostBuffer[0] = threadData->textHostBuffer[0];
	textHostBuffer[1] = threadData->textHostBuffer[1];
	textHostBuffer[2] = threadData->textHostBuffer[2];

	long long unsigned int sourceSpaceSize1 = threadData->sourceSpaceSize1;

	strides_t* stridesSpace1[3];
	stridesSpace1[0] = threadData->stridesSpace1[0];
	stridesSpace1[1] = threadData->stridesSpace1[1];
	stridesSpace1[2] = threadData->stridesSpace1[2];

	strides_t* stridesSpace[1][3];
	stridesSpace[0][0] = stridesSpace1[0];
	stridesSpace[0][1] = stridesSpace1[1];
	stridesSpace[0][2] = stridesSpace1[2];

	firstLastAddr_t* firstLastAddrsSpace1[3];
	firstLastAddrsSpace1[0] = threadData->firstLastAddrsSpace1[0];
	firstLastAddrsSpace1[1] = threadData->firstLastAddrsSpace1[1];
	firstLastAddrsSpace1[2] = threadData->firstLastAddrsSpace1[2];

	firstLastAddr_t* firstLastAddrsSpace[1][3];
	firstLastAddrsSpace[0][0] = firstLastAddrsSpace1[0];
	firstLastAddrsSpace[0][1] = firstLastAddrsSpace1[1];
	firstLastAddrsSpace[0][2] = firstLastAddrsSpace1[2];

	int spaceDataItemSizes[1];
	spaceDataItemSizes[0] = textItemSize;

	char* hostBuffer[1][3];
	hostBuffer[0][0] = textHostBuffer[0];
	hostBuffer[0][1] = textHostBuffer[1];
	hostBuffer[0][2] = textHostBuffer[2];

	int flagGPU[3];
	flagGPU[0] = -1;
	flagGPU[1] = -1;
	flagGPU[2] = -1;
	int flagCPU[3];
	flagCPU[0] = 1;
	flagCPU[1] = 1;
	flagCPU[2] = 1;

	int notDone = 0;

	//mava
	//printf("About entering the while %d\n", myBlock);
	int s = 0;
        //while(notDone < 2)
	while(hipSuccess != hipStreamQuery(*execStream))
	{
		if(flags[myBlock * 12 + s * 2] == flagGPU[s])
		{
			//printf("##########inside if, s is %d\n", s);
			if(notDone == 0)
				endGlobalTimer(myBlock, "@@ prediction");
			else
				endGlobalTimer(myBlock, "@@ computation");

			startGlobalTimer(myBlock); //data assembly

			pthread_t copyThreads[COPYTHREADS - 1];
			copyPackagePattern pkg[COPYTHREADS];
			
			for(int h = 0; h < COPYTHREADS; h ++)
			{
				unsigned int warpChunk = (BLOCKSIZE / WARPSIZE) / COPYTHREADS;
				assert(warpChunk > 0);
				unsigned int warpStart = warpChunk * h;
				unsigned int warpEnd = warpStart + warpChunk;

				pkg[h].tid = h;
				pkg[h].myBlock = myBlock;
				pkg[h].epochDuration = epochDuration;
				pkg[h].warpStart = warpStart;
				pkg[h].warpEnd = warpEnd;
				pkg[h].spaceDataItemSizes[0] = spaceDataItemSizes[0];
				pkg[h].hostBuffer[0] = hostBuffer[0][s];
				pkg[h].srcSpace = fdata;
				pkg[h].stridesSpace[0] = stridesSpace[0][s];
				pkg[h].firstLastAddrsSpace[0] = firstLastAddrsSpace[0][s];
				pkg[h].sourceSpaceSize1 = sourceSpaceSize1;

				if(h < (COPYTHREADS - 1))
					int rc = pthread_create(&copyThreads[h], NULL, copyMethodPattern, (void*) &pkg[h]);
				else
					copyMethodPattern(&pkg[h]);
			}

			for(int h = 0; h < COPYTHREADS - 1; h ++)
				pthread_join(copyThreads[h], NULL);

			endGlobalTimer(myBlock, "@@ Assemble data into pinned buffer");
			
			notDone ++;

			startGlobalTimer(myBlock); //copy from  pinned buffer to GPU memory
			hipMemcpyAsync(&gpuSpaces[0][s][myBlock * threadBlockSize * spaceDataItemSizes[0] * epochDuration], &(hostBuffer[0][s][myBlock * threadBlockSize * epochDuration * spaceDataItemSizes[0]]), epochDuration * threadBlockSize * spaceDataItemSizes[0], hipMemcpyHostToDevice, *streamPtr);

			//while(hipSuccess != hipStreamQuery(*streamPtr));
			endGlobalTimer(myBlock, "@@ Copy from pinned buffer to GPU memory");

			flagCPU[s] *= -1;
			flags[myBlock * 12 + s * 2 + 1] = flagCPU[s];
			flagGPU[s] *= -1;

			asm volatile ("" : : : "memory");
			
			//FIXME: This next hipMemcpyAsync, in an extreme case might take a while to send the signal to GPU, In such case, it may skip one signal@@
			hipMemcpyAsync(&gpuFlags[myBlock * 6 + s], (int*) &flags[myBlock * 12 + s * 2 + 1], sizeof(int), hipMemcpyHostToDevice, *streamPtr);

			//s = (s == 0)? 1 : 0;
			s = (s + 1) % 3;
			startGlobalTimer(myBlock); //computation
		}
	}

	return NULL;
}


int main(int argc, char** argv)
{
	hipError_t errR;
	hipDeviceReset();

	int fd;
	char *fdata;
	struct stat finfo;
	char *fname;

	if (argc < 2)
	{
		printf("USAGE: %s <inputfilename>\n", argv[0]);
		exit(1);
	}

	fname = argv[1];
	fd = open(fname, O_RDONLY);
	fstat(fd, &finfo);
	printf("Allocating %lluMB for the input file.\n", ((long long unsigned int)finfo.st_size) / (1 << 20));
	fdata = (char *) malloc(finfo.st_size);
	size_t fileSize = (size_t) finfo.st_size;

	largeInt maxReadSize = GB;
	largeInt readed = 0;
	largeInt toRead = 0;

	if(fileSize > maxReadSize)
        {
                largeInt offset = 0;
                while(offset < fileSize)
                {
                        toRead = (maxReadSize < (fileSize - offset))? maxReadSize : (fileSize - offset);
                        readed += pread(fd, fdata + offset, toRead, offset);
                        printf("writing %lliMB\n", toRead / (1 << 20));
                        //pwrite(fdw, fdata + offset, toRead, offset);
                        offset += toRead;
                }
        }
        else
                readed = read (fd, fdata, fileSize);


	if(readed != fileSize)
		printf("Not all of the file is read\n");

	dim3 block(BLOCKSIZE, 1, 1);
	dim3 block2((BLOCKSIZE * 2), 1, 1);
	dim3 grid(MAXBLOCKS, 1, 1);
	int numThreads = BLOCKSIZE * grid.x * grid.y;

	int numRecords = fileSize / RECORD_SIZE;

	//======================================================//
	int chunkSize = EPOCHCHUNK * (1 << 20);

	//TODO: make epochNum unnecessary
	int epochNum = (int) (fileSize / chunkSize);
	if(fileSize % chunkSize)
		epochNum ++;

	printf("Number of epochs: %d\n", epochNum);
	//======================================================//

	//=================== Max num of iterations ============//
	int maxIterations = numRecords / numThreads;

	maxIterations ++;
	if(epochNum > 1)
		maxIterations /= (epochNum);
	//======================================================//


	int iterations = maxIterations;
	if(iterations % 8 != 0)
		iterations += (8 - (iterations % 8));

	//========= URLAddrHostBuffer ===========//
	unsigned int textAddrsHostBufferSize = sizeof(ptr_t) * iterations * numThreads * 3;
	ptr_t* tempTextAddrHostBuffer;
	tempTextAddrHostBuffer = (ptr_t*) malloc(textAddrsHostBufferSize + MEMORY_ALIGNMENT);
	ptr_t* hostTextAddrHostBuffer;
	hostTextAddrHostBuffer = (ptr_t*) ALIGN_UP(tempTextAddrHostBuffer, MEMORY_ALIGNMENT);
	memset((void*) hostTextAddrHostBuffer, 0, textAddrsHostBufferSize);
	hipHostRegister((void*) hostTextAddrHostBuffer, textAddrsHostBufferSize, hipHostMallocMapped);
	ptr_t* textAddrs;
	hipHostGetDevicePointer((void **)&textAddrs, (void *)hostTextAddrHostBuffer, 0);
	//============================================//	

	//========= URLHostBuffer ===========//
	int textHostBufferSize = RECORD_SIZE * iterations * numThreads * 3;
	char* tempTextHostBuffer;
	tempTextHostBuffer = (char*) malloc(textHostBufferSize + MEMORY_ALIGNMENT);
	char* hostTextHostBuffer;
	hostTextHostBuffer = (char*) ALIGN_UP(tempTextHostBuffer, MEMORY_ALIGNMENT);
	memset((void*) hostTextHostBuffer, 0, textHostBufferSize);
	hipHostRegister((void*) hostTextHostBuffer, textHostBufferSize, hipHostMallocMapped);
	//============================================//	


	//================= completeFlag ===============//
	int flagSize = grid.x * grid.y * 12 * sizeof(int);
	int volatile * volatile tempCompleteFlag = (int*) malloc(flagSize + MEMORY_ALIGNMENT);
	int volatile * volatile hostCompleteFlag = (int*) ALIGN_UP(tempCompleteFlag, MEMORY_ALIGNMENT);
	memset((void*) hostCompleteFlag, 0, flagSize);
	hipHostRegister((void*) hostCompleteFlag, flagSize, hipHostMallocMapped);
	int volatile * volatile flags;
	hipHostGetDevicePointer((void **)&flags, (void *)hostCompleteFlag, 0);

	int* gpuFlags;
	hipMalloc((void**) &gpuFlags, flagSize / 2);
	hipMemset(gpuFlags, 0, flagSize / 2);
	//============================================//

	//================= strides ===============//
	int stridesSize = numThreads * sizeof(strides_t) * 3;
	strides_t* tempStridesSpace1;
	tempStridesSpace1 = (strides_t*) malloc(stridesSize + MEMORY_ALIGNMENT);
	strides_t* hostStridesSpace1;
	hostStridesSpace1 = (strides_t*) ALIGN_UP(tempStridesSpace1, MEMORY_ALIGNMENT);
	memset((void*) hostStridesSpace1, 0, stridesSize);
	hipHostRegister((void*) hostStridesSpace1, stridesSize, hipHostMallocMapped);
	strides_t* stridesSpace1;
	hipHostGetDevicePointer((void **)&stridesSpace1, (void *)hostStridesSpace1, 0);
	//============================================//

	//================= firstLastAddrs ===============//
	int fistLastAddrSize = numThreads * sizeof(firstLastAddr_t) * 3;//16 * sizeof(int) + 2 * sizeof(long long int);
	firstLastAddr_t* tempFirstLastSpace1;
	tempFirstLastSpace1 = (firstLastAddr_t*) malloc(fistLastAddrSize + MEMORY_ALIGNMENT);
	firstLastAddr_t* hostFirstLastSpace1;
	hostFirstLastSpace1 = (firstLastAddr_t*) ALIGN_UP(tempFirstLastSpace1, MEMORY_ALIGNMENT);
	memset((void*) hostFirstLastSpace1, 0, fistLastAddrSize);
	hipHostRegister((void*) hostFirstLastSpace1, fistLastAddrSize, hipHostMallocMapped);
	firstLastAddr_t* firstLastAddrsSpace1;
	hipHostGetDevicePointer((void **)&firstLastAddrsSpace1, (void *)hostFirstLastSpace1, 0);
	//============================================//

	char* textData;
	hipMalloc((void**) &textData, RECORD_SIZE * iterations * numThreads * 3);

	char* phony = (char*) 0x0;
	hipStream_t execStream;
	hipStreamCreate(&execStream);
	hipStream_t copyStream;
	hipStreamCreate(&copyStream);
	
	//============ initializing the hash table and page table ==================//
	multipassConfig_t* mbk = initMultipassBookkeeping(	(int*) hostCompleteFlag, 
								gpuFlags, 
								flagSize,
								GROUP_SIZE, 
								numThreads,
								epochNum,
								numRecords);

	//==========================================================================//

	struct timeval partial_start, partial_end, bookkeeping_start, bookkeeping_end, passtime_start, passtime_end;
	time_t sec;
	time_t ms;
	time_t diff;

	errR = hipGetLastError();
	printf("#######Error before calling the kernel is: %s\n", hipGetErrorString(errR));

	gettimeofday(&partial_start, NULL);
	int passNo = 1;
	bool failedFlag = false;
	do
	{
		printf("====================== starting pass %d ======================\n", passNo);
		gettimeofday(&passtime_start, NULL);

		wordCountKernelMultipass<<<grid, block2, 0, execStream>>>(
				phony, 
				numRecords, //TODO fill this
				textAddrs,
				textData,
				flags,
				gpuFlags,
				stridesSpace1,
				firstLastAddrsSpace1,
				iterations,
				epochNum,
				mbk->dmyNumbers,
				numThreads,
				mbk->dpconfig,
				mbk->dhconfig,
				mbk->dstates,
				mbk->dfailedFlag,
				mbk->depochSuccessStatus
				);


		pthread_t threads[MAXBLOCKS];
		dataPackage* argument[MAXBLOCKS];

		for(int m = 0; m < MAXBLOCKS; m ++)
		{
			startGlobalTimer(m);  //prediction
			argument[m] = (dataPackage*) malloc(sizeof(dataPackage));
			argument[m]->streamPtr = &copyStream;
			argument[m]->execStream = &execStream;
			argument[m]->flags = hostCompleteFlag;
			argument[m]->gpuFlags = gpuFlags;
			argument[m]->fdata = fdata;
			argument[m]->myBlock = m;
			argument[m]->threadBlockSize = BLOCKSIZE;
			argument[m]->textItems = iterations;
			argument[m]->textHostBuffer[0] = hostTextHostBuffer;
			argument[m]->textHostBuffer[1] = hostTextHostBuffer + iterations * numThreads * RECORD_SIZE;
			argument[m]->textHostBuffer[2] = hostTextHostBuffer + iterations * numThreads * RECORD_SIZE * 2;
			argument[m]->textAddrs[0] = hostTextAddrHostBuffer;
			argument[m]->textAddrs[1] = hostTextAddrHostBuffer + iterations * numThreads;
			argument[m]->textAddrs[2] = hostTextAddrHostBuffer +  iterations * numThreads * 2;
			argument[m]->textData[0] = textData;
			argument[m]->textData[1] = textData + iterations * numThreads * RECORD_SIZE;
			argument[m]->textData[2] = textData +  iterations * numThreads * RECORD_SIZE * 2;
			argument[m]->stridesSpace1[0] = hostStridesSpace1;
			argument[m]->stridesSpace1[1] = hostStridesSpace1 + numThreads;
			argument[m]->stridesSpace1[2] = hostStridesSpace1 + numThreads * 2;
			argument[m]->epochDuration = iterations;
			argument[m]->firstLastAddrsSpace1[0] = hostFirstLastSpace1;
			argument[m]->firstLastAddrsSpace1[1] = hostFirstLastSpace1 + numThreads;
			argument[m]->firstLastAddrsSpace1[2] = hostFirstLastSpace1 + numThreads * 2;
			argument[m]->textItemSize = RECORD_SIZE;
			argument[m]->sourceSpaceSize1 = fileSize;

			pthread_create(&threads[m], NULL, pipelineData, (void*) argument[m]);
		}


		while(hipSuccess != hipStreamQuery(execStream))
			usleep(300);	


		errR = hipGetLastError();
		printf("Error after calling the kernel is: %s\n", hipGetErrorString(errR));

		hipDeviceSynchronize();

		gettimeofday(&passtime_end, NULL);
		sec = passtime_end.tv_sec - passtime_start.tv_sec;
		ms = passtime_end.tv_usec - passtime_start.tv_usec;
		diff = sec * 1000000 + ms;
		printf("\n%10s:\t\t%0.1fms\n", "Pass time ", (double)((double)diff/1000.0));

		for(int m = 0; m < MAXBLOCKS; m ++)
			endGlobalTimer(m, "@@ computation");



		//======================= Some reseting ===========================
		

		gettimeofday(&bookkeeping_start, NULL);
		failedFlag = checkAndResetPass(mbk);
		
		
		gettimeofday(&bookkeeping_end, NULL);
		sec = bookkeeping_end.tv_sec - bookkeeping_start.tv_sec;
		ms = bookkeeping_end.tv_usec - bookkeeping_start.tv_usec;
		diff = sec * 1000000 + ms;
		printf("\n%10s:\t\t%0.1fms\n", "Pass bookkeeping", (double)((double)diff/1000.0));


		passNo ++;


	} while(failedFlag);

	gettimeofday(&partial_end, NULL);
	sec = partial_end.tv_sec - partial_start.tv_sec;
	ms = partial_end.tv_usec - partial_start.tv_usec;
	diff = sec * 1000000 + ms;
	printf("\n%10s:\t\t%0.1fms\n", "Total time", (double)((double)diff/1000.0));

	
	bucketGroup_t* groups = (bucketGroup_t*) malloc(mbk->numGroups * sizeof(bucketGroup_t));
	hipMemcpy(groups, mbk->hconfig->groups, mbk->numGroups * sizeof(bucketGroup_t), hipMemcpyDeviceToHost);

#ifdef DISPLAY_RESULTS
	int tabCount = 0;
	for(int i = 0; i < 10; i ++)
	{
		for(int j = 0; j < 5; j ++)
		{
			hashBucket_t* bucket = groups[i].buckets[j];

			while(bucket != NULL)
			{
				char* url = (char*) getKey(bucket);

				for(int k = 0; k < tabCount; k ++)
					printf("\t");
				printf("URL: ");
				for(int m = 0; m < bucket->keySize; m ++)
					printf("%c", url[m]);

				int* value = (int*) getValue(bucket);
				printf(": %d\n", *value);
				bucket = bucket->next;

				tabCount ++;
			}
			tabCount = 0;
		}
		
	
	}
#endif

	int totalDepth = 0;
	int totalValidBuckets = 0;
	int totalEmpty = 0;
	int maximumDepth = 0;
	for(int i = 0; i < mbk->numGroups; i ++)
	{
		for(int j = 0; j < GROUP_SIZE; j ++)
		{
			hashBucket_t* bucket = groups[i].buckets[j];
			if(bucket == NULL)
				totalEmpty ++;
			else
				totalValidBuckets ++;

			int localMaxDepth = 0;
			while(bucket != NULL)
			{
				totalDepth ++;
				localMaxDepth ++;
				bucket = bucket->next;
			}
			if(localMaxDepth > maximumDepth)
				maximumDepth = localMaxDepth;
		}
	
	}

	float emptyPercentage = ((float) totalEmpty / (float) NUM_BUCKETS) * (float) 100;
	float averageDepth = (float) totalDepth / (float) totalValidBuckets;
	printf("Empty percentage: %0.1f\n", emptyPercentage);
	printf("Average depth: %0.1f\n", averageDepth);
	printf("Max depth: %d\n", maximumDepth);

	return 0;
}
