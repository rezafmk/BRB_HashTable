#include "hip/hip_runtime.h"
#include "hashGlobal.h"

void hashtableInit(int numBuckets, hashtableConfig_t* hconfig)
{
	hconfig->numBuckets = numBuckets;
	int numGroups = (numBuckets + (GROUP_SIZE - 1)) / GROUP_SIZE;
	hipMalloc((void**) &(hconfig->groups), numGroups * sizeof(bucketGroup_t));
	hipMemset(hconfig->groups, 0, numGroups * sizeof(bucketGroup_t));
	//hconfig->groups = (bucketGroup_t*) malloc(numGroups * sizeof(bucketGroup_t));
}

__device__ unsigned int hashFunc(char* str, int len, unsigned numBuckets)
{
        unsigned hash = 2166136261;
        unsigned FNVMultiple = 16777619;

        for(int i = 0; i < len; i ++)
        {
                char c = str[i];

                hash += (int) c;
                hash = hash * FNVMultiple;  /* multiply by the magic number */
                hash += len;
                hash -= (int) c;
        }

        return hash % numBuckets;
}


__device__ void resolveSameKeyAddition(void const* key, void* value, void* oldValue)
{
	*((int*) oldValue) += 1;
}

__device__ hashBucket_t* containsKey(hashBucket_t* bucket, void* key, int keySize)
{
	while(bucket != NULL)
	{
		char* oldKey = (char*) ((largeInt) bucket + sizeof(hashBucket_t));
		bool success = true;
		//OPTIMIZE: do the comparisons 8-byte by 8-byte
		for(int i = 0; i < keySize; i ++)
		{
			if(oldKey[i] != ((char*) key)[i])
			{
				success = false;
				break;
			}
		}
		if(success)
			break;

		if(bucket->isNextDead == 0)
			bucket = bucket->next;
		else
			bucket = NULL;
	}

	return bucket;
}

__device__ bool atomicAttemptIncRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	bool success;
	do
	{
		success = false;
		assume = oldRefCount;
		if(oldRefCount >= 0)
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, oldRefCount + 1);
			success = true;
		}
	} while(oldRefCount != assume);

	return success;
}

__device__ int atomicDecRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	do
	{
		assume = oldRefCount;
		if(oldRefCount >= 0) // During normal times
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, (unsigned) (oldRefCount - 1));
		}
		else // During failure
		{
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, (unsigned) (oldRefCount + 1));
		}

	} while(oldRefCount != assume);

	return oldRefCount;
}

__device__ bool atomicNegateRefCount(int* refCount)
{
	int oldRefCount = *refCount;
	int assume;
	do
	{
		assume = oldRefCount;
		if(oldRefCount >= 0)
			oldRefCount = (int) atomicCAS((unsigned*) refCount, (unsigned) oldRefCount, ((oldRefCount * (-1)) - 1));

	} while(oldRefCount != assume);

	return (oldRefCount >= 0);
	
}

__device__ bool addToHashtable(void* key, int keySize, void* value, int valueSize, hashtableConfig_t* hconfig, pagingConfig_t* pconfig)
{
	bool success = true;
	unsigned hashValue = hashFunc((char*) key, keySize, hconfig->numBuckets);

	unsigned groupNo = hashValue / GROUP_SIZE;
	unsigned offsetWithinGroup = hashValue % GROUP_SIZE;

	bucketGroup_t* group = &(hconfig->groups[groupNo]);
	
	hashBucket_t* existingBucket;

	int keySizeAligned = (keySize % ALIGNMET == 0)? keySize : keySize + (ALIGNMET - (keySize % ALIGNMET));
	int valueSizeAligned = (valueSize % ALIGNMET == 0)? valueSize : valueSize + (ALIGNMET - (valueSize % ALIGNMET));

	unsigned oldLock = 1;

	do
	{
		oldLock = atomicExch((unsigned*) &(group->locks[offsetWithinGroup]), 1);

		if(oldLock == 0)
		{
			hashBucket_t* bucket = NULL;
			if(group->buckets[offsetWithinGroup] != NULL)
				bucket = (hashBucket_t*) ((largeInt) group->buckets[offsetWithinGroup] - pconfig->hashTableOffset + (largeInt) pconfig->dbuffer);
			//First see if the key already exists in one of the entries of this bucket
			//The returned bucket is the 'entry' in which the key exists
			if(group->isNextDead[offsetWithinGroup] != 1 && (existingBucket = containsKey(bucket, key, keySize)) != NULL)
			{
				void* oldValue = (void*) ((largeInt) existingBucket + sizeof(hashBucket_t) + keySizeAligned);
				resolveSameKeyAddition(key, value, oldValue);
			}
			else
			{
				hashBucket_t* newBucket = (hashBucket_t*) multipassMalloc(sizeof(hashBucket_t) + keySizeAligned + valueSizeAligned, group, pconfig, groupNo);
				if(newBucket != NULL)
				{
					//TODO reduce the base offset if not null
					//newBucket->next = (bucket == NULL)? NULL : (hashBucket_t*) ((largeInt) bucket - (largeInt) pconfig->dbuffer);
					//group->failed = 1;
					newBucket->next = NULL;
					if(bucket != NULL)
						newBucket->next = (hashBucket_t*) ((largeInt) bucket - (largeInt) pconfig->dbuffer + pconfig->hashTableOffset);
					if(group->isNextDead[offsetWithinGroup] == 1)
						newBucket->isNextDead = 1;
						
					group->buckets[offsetWithinGroup] = (hashBucket_t*) ((largeInt) newBucket - (largeInt) pconfig->dbuffer + pconfig->hashTableOffset);
					group->isNextDead[offsetWithinGroup] = 0;

					//TODO: this assumes that input key is aligned by ALIGNMENT, which is not a safe assumption
					for(int i = 0; i < (keySizeAligned / ALIGNMET); i ++)
						*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) key + i * ALIGNMET));
					for(int i = 0; i < (valueSizeAligned / ALIGNMET); i ++)
						*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned + i * ALIGNMET)) = *((largeInt*) ((largeInt) value + i * ALIGNMET));
				}
				else
				{
					success = false;
				}
			}

			atomicExch((unsigned*) &(group->locks[offsetWithinGroup]), 0);
		}
	} while(oldLock == 1);

	return success;
}


