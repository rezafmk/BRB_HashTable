#include "hip/hip_runtime.h"
#include "hashGlobal.h"
#include "global.h"

void initPaging(largeInt availableGPUMemory, multipassConfig_t* mbk)
{

	mbk->totalNumPages = availableGPUMemory / PAGE_SIZE;
	printf("@INFO: total number of pages: %d [each %dKB]\n", mbk->totalNumPages, (PAGE_SIZE / (1 << 10)));
	mbk->initialPageAssignedCounter = 0;
	mbk->initialPageAssignedCap = mbk->totalNumPages;

	hipMalloc((void**) &(mbk->dbuffer), mbk->totalNumPages * PAGE_SIZE);
	hipMemset(mbk->dbuffer, 0, mbk->totalNumPages * PAGE_SIZE);
	printf("@INFO: done allocating base buffer in GPU memory\n");

	//This has to be allocated GPU-side
	mbk->hpages = (page_t*) malloc(mbk->totalNumPages * sizeof(page_t));
	for(int i = 0; i < mbk->totalNumPages; i ++)
	{
		mbk->hpages[i].id = i;
		mbk->hpages[i].next = NULL;
		mbk->hpages[i].used = 0;
	}
	printf("@INFO: done initializing pages meta data\n");
	hipMalloc((void**) &(mbk->pages), mbk->totalNumPages * sizeof(page_t));
	hipMemcpy(mbk->pages, mbk->hpages, mbk->totalNumPages * sizeof(page_t), hipMemcpyHostToDevice);

	printf("@INFO: done doing initPaging\n");
}



//TODO: currently we don't mark a bucket group to not ask for more memory if it previously revoked its pages
__device__ void* multipassMalloc(unsigned size, bucketGroup_t* myGroup, multipassConfig_t* mbk, int groupNo)
{
	page_t* parentPage = myGroup->parentPage;

	unsigned oldUsed = 0;
	if(parentPage != NULL)
	{
		oldUsed = atomicAdd(&(parentPage->used), size);
		if((oldUsed + size) < PAGE_SIZE)
		{
			return (void*) ((largeInt) mbk->dbuffer + parentPage->id * PAGE_SIZE + oldUsed);
		}
	}

	page_t* newPage = NULL;
	//acquire some lock
	unsigned oldLock = 1;
	do
	{
		oldLock = atomicExch(&(myGroup->pageLock), 1);

		if(oldLock == 0)
		{
			//Re-testing if the parent page has room (because the partenPage might have changed)
			parentPage = myGroup->parentPage;
			if(parentPage != NULL)
			{
				oldUsed = atomicAdd(&(parentPage->used), size);
				if((oldUsed + size) < PAGE_SIZE)
				{
					//Unlocking
					atomicExch(&(myGroup->pageLock), 0);
					return (void*) ((largeInt) mbk->dbuffer + parentPage->id * PAGE_SIZE + oldUsed);
				}
			}
			
			newPage = allocateNewPage(mbk, groupNo);

			//If no more page exists and no page is used yet (for this bucketgroup), don't do anything
			if(newPage == NULL)
			{
				//releaseLock
				atomicExch(&(myGroup->pageLock), 0);
				return NULL;
			}

			newPage->next = parentPage;
			myGroup->parentPage = newPage;

			//Unlocking
			atomicExch(&(myGroup->pageLock), 0);
		}

	} while(oldLock == 1);

	//This assumes that the newPage is not already full, which is to be tested.
	oldUsed = atomicAdd(&(newPage->used), size);

	if((oldUsed + size) < PAGE_SIZE)
		return (void*) ((largeInt) mbk->dbuffer + oldUsed + newPage->id * PAGE_SIZE);
	else
	{
		return NULL;
	}
}

__device__ page_t* allocateNewPage(multipassConfig_t* mbk, int groupNo)
{
	int pageIdToAllocate = atomicInc((unsigned*) &(mbk->initialPageAssignedCounter), INT_MAX);
	if(pageIdToAllocate < mbk->totalNumPages)
	{
		return &(mbk->pages[pageIdToAllocate]);
	}
	return NULL;
}


