#include "hip/hip_runtime.h"
#include "hashGlobal.h"
#define STATISTICS 1

void hashtableInit(unsigned numBuckets, multipassConfig_t* mbk, unsigned groupSize)
{
	mbk->numBuckets = numBuckets;
	mbk->groupSize = groupSize;
	int numGroups = (numBuckets + (groupSize - 1)) / groupSize;
	hipMalloc((void**) &(mbk->groups), numGroups * sizeof(bucketGroup_t));
	hipMalloc((void**) &(mbk->buckets), numBuckets * sizeof(hashBucket_t*));
	hipMalloc((void**) &(mbk->dbuckets), numBuckets * sizeof(hashBucket_t*));
	hipMalloc((void**) &(mbk->locks), numBuckets * sizeof(unsigned));
	hipMalloc((void**) &(mbk->isNextDeads), numBuckets * sizeof(short));
	
	hipMemset(mbk->groups, 0, numGroups * sizeof(bucketGroup_t));
	hipMemset(mbk->dbuckets, 0, numBuckets * sizeof(hashBucket_t*));
	hipMemset(mbk->buckets, 0, numBuckets * sizeof(hashBucket_t*));
	hipMemset(mbk->locks, 0, numBuckets * sizeof(unsigned));
	hipMemset(mbk->isNextDeads, 0, numBuckets * sizeof(short));
}


__device__ unsigned int hashFunc(char* str, int len, unsigned numBuckets)
{
        largeInt number = ((input_t*) str)->data1;
        return number % numBuckets;
}

__device__ bool addNewValueAtomically(void const* key, void* value, int valueSize, void* oldValue, bucketGroup_t* group, multipassConfig_t* mbk)
{
	//TODO: here make it compatible with the new structure of vlaue at the end fo the bucket...
	valueHolder_t* newValue = (valueHolder_t*) multipassMallocValue(sizeof(valueHolder_t) + valueSize, group, mbk);
	if(newValue != NULL)
	{
		newValue->valueSize = (largeInt) valueSize;
		setValue(newValue, value, valueSize);

		largeInt atomicReturnedValue, previousValue;
		largeInt toBeInsertedValue = ((largeInt) newValue - (largeInt) mbk->dbuffer + group->valueParentPage->hashTableOffset); 

		newValue->next = ((valueHolder_t*) oldValue)->next;

		// Appending the new value at the beginning of the linked list atomically
		do
		{

			previousValue = (largeInt) ((valueHolder_t*) oldValue)->next;
			newValue->next = ((valueHolder_t*) oldValue)->next;

			atomicReturnedValue = atomicCAS((unsigned long long int*) &(((valueHolder_t*) oldValue)->next), previousValue, toBeInsertedValue);

		} while(previousValue != atomicReturnedValue);

		return true;
	}
	return false;
}

__device__ hashBucket_t* containsKeyUntilEntry(hashBucket_t* bucket, void* key, int keySize, hashBucket_t* lastCheckedBucket, multipassConfig_t* mbk)
{
	// This should never be null... that's why I'm not checking for null
	while(bucket != lastCheckedBucket)
	{
		char* oldKey = (char*) ((largeInt) bucket + sizeof(hashBucket_t));
		bool success = true;

		int i = 0;
		for(; i < keySize/ALIGNMET && success; i ++)
		{
			if(((largeInt*) oldKey)[i] != ((largeInt*) key)[i])
				success = false;
		}
		i *= ALIGNMET;
		for(; i < keySize && success; i ++)
		{
			if(oldKey[i] != ((char*) key)[i])
				success = false;
		}

		if(success)
		{
			return bucket;
		}

		bucket = bucket->dnext;
	}
	return NULL;
}


__device__ hashBucket_t* containsKey(hashBucket_t* bucket, void* key, int keySize, multipassConfig_t* mbk)
{
	while(bucket != NULL)
	{
		char* oldKey = (char*) ((largeInt) bucket + sizeof(hashBucket_t));
		bool success = true;

		int i = 0;
		for(; i < keySize/ALIGNMET && success; i ++)
		{
			if(((largeInt*) oldKey)[i] != ((largeInt*) key)[i])
				success = false;
		}
		i *= ALIGNMET;
		for(; i < keySize && success; i ++)
		{
			if(oldKey[i] != ((char*) key)[i])
				success = false;
		}


		if(success)
			break;

		if(bucket->isNextDead == 0 && bucket->dnext != NULL)
			bucket = bucket->dnext;
		else
			bucket = NULL;
	}

	return bucket;
}

__device__ bool insert_multi_value(void* key, int keySize, void* value, int valueSize, multipassConfig_t* mbk)
{
	bool success = true;
	unsigned hashValue = hashFunc((char*) key, keySize, mbk->numBuckets);

	unsigned groupNo = hashValue / mbk->groupSize;

	bucketGroup_t* group = &(mbk->groups[groupNo]);
	if(group->overflownKey == 1 && group->overflownValue == 1)
		return false;
	
	hashBucket_t* existingBucket;
	hashBucket_t* lastCheckedBucket;

	int keySizeAligned = (keySize % ALIGNMET == 0)? keySize : keySize + (ALIGNMET - (keySize % ALIGNMET));
	int valueSizeAligned = (valueSize % ALIGNMET == 0)? valueSize : valueSize + (ALIGNMET - (valueSize % ALIGNMET));

	hashBucket_t* dbucket = mbk->dbuckets[hashValue];
	lastCheckedBucket = dbucket;
	

	// Scenario 1: if the key already exists, without acquiring the lock, just insert the new value atomically
	if(mbk->isNextDeads[hashValue] != 1 && (existingBucket = containsKey(dbucket, key, keySize, mbk)) != NULL)
	{
		void* oldValue = (void*) ((largeInt) existingBucket + sizeof(hashBucket_t) + keySizeAligned);
		if(!addNewValueAtomically(key, value, valueSizeAligned, oldValue, group, mbk))
		{
			group->needed = 1;
			page_t* temp = group->parentPage;
			while(temp != NULL)
			{
				temp->needed = 1;
				temp = temp->next;
			}
			success = false;
		}
#ifdef STATISTICS
		atomicInc((unsigned*) &(mbk->counter1), INT_MAX);
#endif
	}
	else 
	{
		unsigned oldLock = 1;

		do
		{
			oldLock = atomicExch((unsigned*) &(mbk->locks[hashValue]), 1);
			if(oldLock == 0)
			{
				dbucket = mbk->dbuckets[hashValue];

				if(mbk->isNextDeads[hashValue] != 1 && (existingBucket = containsKeyUntilEntry(dbucket, key, keySize, lastCheckedBucket, mbk)) != NULL)
				{
					// Scenario 2: the key is added a moment ago (just before we acquired the lock), let's unlock and inser the value atomically
					atomicExch((unsigned*) &(mbk->locks[hashValue]), 0);

					void* oldValue = (void*) ((largeInt) existingBucket + sizeof(hashBucket_t) + keySizeAligned);
					if(!addNewValueAtomically(key, value, valueSizeAligned, oldValue, group, mbk))
					{
						group->needed = 1;
						page_t* temp = group->parentPage;
						while(temp != NULL)
						{
							temp->needed = 1;
							temp = temp->next;
						}
						success = false;
					}
#ifdef STATISTICS
					atomicInc((unsigned*) &(mbk->counter2), INT_MAX);
#endif

				}
				else
				{
#ifdef STATISTICS
					atomicInc((unsigned*) &(mbk->counter3), INT_MAX);
#endif
					hashBucket_t* newBucket = (hashBucket_t*) multipassMalloc(sizeof(hashBucket_t) + keySizeAligned + sizeof(valueHolder_t) + valueSizeAligned, group, mbk);
					hashBucket_t* hbucket = mbk->buckets[hashValue];

					if(newBucket != NULL)
					{
						//TODO reduce the base offset if not null
						//newBucket->next = (bucket == NULL)? NULL : (hashBucket_t*) ((largeInt) bucket - (largeInt) mbk->dbuffer);
						newBucket->dnext = NULL;
						newBucket->next = NULL;
						if(dbucket != NULL)
						{
							newBucket->dnext = dbucket;
							newBucket->next = hbucket;
						}

						if(mbk->isNextDeads[hashValue] == 1)
							newBucket->isNextDead = 1;
						newBucket->keySize = (short) keySize;
						newBucket->valueSize = (short) valueSize;
						newBucket->dvalueHolder = (valueHolder_t*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned);
						newBucket->valueHolder = (valueHolder_t*) ((largeInt) newBucket->dvalueHolder - (largeInt) mbk->dbuffer + group->parentPage->hashTableOffset);;


						mbk->isNextDeads[hashValue] = 0;

						for(int i = 0; i < (keySizeAligned / ALIGNMET); i ++)
							*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) key + i * ALIGNMET));
						setValue(newBucket->dvalueHolder, value, valueSizeAligned);
						newBucket->dvalueHolder->next = NULL;
						newBucket->dvalueHolder->valueSize = (largeInt) valueSize;

						for(int i = 0; i < (valueSizeAligned / ALIGNMET); i ++)
							*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned + sizeof(valueHolder_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) value + i * ALIGNMET));
						((valueHolder_t*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned))->next = NULL;
						((valueHolder_t*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned))->valueSize = valueSize;

						mbk->dbuckets[hashValue] = newBucket;
						mbk->buckets[hashValue] = (hashBucket_t*) ((largeInt) newBucket - (largeInt) mbk->dbuffer + group->parentPage->hashTableOffset);

					}
					else
					{
						success = false;
					}

					atomicExch((unsigned*) &(mbk->locks[hashValue]), 0);

				}

			}
		} while(oldLock == 1);
	
		
	}

	return success;
}

__device__ bool insert_basic(void* key, int keySize, void* value, int valueSize, multipassConfig_t* mbk)
{
	bool success = true;
	unsigned hashValue = hashFunc((char*) key, keySize, mbk->numBuckets);

	unsigned groupNo = hashValue / mbk->groupSize;
	//unsigned groupNo = hashValue / GROUP_SIZE;

	bucketGroup_t* group = &(mbk->groups[groupNo]);
	
	int keySizeAligned = (keySize % ALIGNMET == 0)? keySize : keySize + (ALIGNMET - (keySize % ALIGNMET));
	int valueSizeAligned = (valueSize % ALIGNMET == 0)? valueSize : valueSize + (ALIGNMET - (valueSize % ALIGNMET));

	hashBucket_t* newBucket = (hashBucket_t*) multipassMalloc(sizeof(hashBucket_t) + keySizeAligned + sizeof(valueHolder_t) + valueSizeAligned, group, mbk);
	if(newBucket != NULL)
	{
		//TODO reduce the base offset if not null
		//newBucket->next = (bucket == NULL)? NULL : (hashBucket_t*) ((largeInt) bucket - (largeInt) mbk->dbuffer);
		newBucket->next = NULL;
		newBucket->keySize = (short) keySize;
		newBucket->valueSize = (short) valueSize;

		for(int i = 0; i < (keySizeAligned / ALIGNMET); i ++)
			*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) key + i * ALIGNMET));
		for(int i = 0; i < (valueSizeAligned / ALIGNMET); i ++)
			*((largeInt*) ((largeInt) newBucket + sizeof(hashBucket_t) + keySizeAligned + i * ALIGNMET)) = *((largeInt*) ((largeInt) value + i * ALIGNMET));

		largeInt atomicOldValue, oldValue;
		largeInt newValue = ((largeInt) newBucket - (largeInt) mbk->dbuffer + mbk->hashTableOffset);

		do
		{
			newBucket->isNextDead = 0;
			if(mbk->isNextDeads[hashValue] == 1)
				newBucket->isNextDead = 1;
			oldValue = (largeInt) mbk->buckets[hashValue];
			newBucket->next = (hashBucket_t*) oldValue;
			atomicOldValue = atomicCAS((unsigned long long int*) &(mbk->buckets[hashValue]), oldValue, newValue);
		} while(oldValue != atomicOldValue);
	}
	else
	{
		success = false;
	}

	return success;
}

__device__ hashBucket_t* lookup_basic(void* key, int keySize, multipassConfig_t* mbk)
{
	unsigned hashValue = hashFunc((char*) key, keySize, mbk->numBuckets);

	unsigned groupNo = hashValue / mbk->groupSize;
	//unsigned groupNo = hashValue / GROUP_SIZE;

	bucketGroup_t* group = &(mbk->groups[groupNo]);

	int isCPUResident = mbk->isNextDeads[hashValue];
	hashBucket_t* bucket = mbk->buckets[hashValue];

	while(bucket != NULL)
	{
		if(isCPUResident == 0)
		{
			bucket = (hashBucket_t*) ((largeInt) bucket - mbk->hashTableOffset + (largeInt) mbk->dbuffer);
		}

		
		char* oldKey = (char*) ((largeInt) bucket + sizeof(hashBucket_t));
		bool success = true;
		int i = 0;
		for(; i < keySize/ALIGNMET && success; i ++)
		{
			if(((largeInt*) oldKey)[i] != ((largeInt*) key)[i])
				success = false;
		}
		i *= ALIGNMET;
		for(; i < keySize && success; i ++)
		{
			if(oldKey[i] != ((char*) key)[i])
				success = false;
		}

		if(success)
			return bucket;
		
		isCPUResident = bucket->isNextDead;
		bucket = bucket->next;
	}

	return NULL;
}

__global__ void setGroupsPointersDead(multipassConfig_t* mbk, unsigned numBuckets)
{
	int index = TID;
	if(index < numBuckets)
	{
		int groupNo = index / mbk->groupSize;
		if(mbk->groups[groupNo].needed == 0)
			mbk->isNextDeads[index] = 1;
	}
	if(index < mbk->groupSize)
	{
		mbk->groups[index].overflownKey = 0;
		mbk->groups[index].overflownValue = 0;
	}

}



multipassConfig_t* initMultipassBookkeeping(	int numThreads,
						int numRecords,
						int pagePerGroup)
{
	
	multipassConfig_t* mbk = (multipassConfig_t*) malloc(sizeof(multipassConfig_t));
	mbk->numThreads = numThreads;
	mbk->numRecords = numRecords;


	mbk->availableGPUMemory = (1400 * (1 << 20));
	mbk->hhashTableBufferSize = MAX_NO_PASSES * mbk->availableGPUMemory;
	mbk->hhashTableBaseAddr = malloc(mbk->hhashTableBufferSize);
	memset(mbk->hhashTableBaseAddr, 0, mbk->hhashTableBufferSize);
	mbk->hashTableOffset = (largeInt) mbk->hhashTableBaseAddr;

	//This is how we decide the number of groups: based on the number of available pages, we make sure 
	//group size is calculated in a way that a given number of `pagePerGroup` pages are assigned to each group
	int availableNumPages = mbk->availableGPUMemory / PAGE_SIZE;
	mbk->groupSize = (pagePerGroup * NUM_BUCKETS) / availableNumPages;
	mbk->numGroups = (NUM_BUCKETS + (mbk->groupSize - 1)) / mbk->groupSize;
	//mbk->numGroups = (NUM_BUCKETS + (GROUP_SIZE - 1)) / GROUP_SIZE;
	printf("############# groupSize: %d, number of groups: %d\n", mbk->groupSize, mbk->numGroups);


	hipMalloc((void**) &(mbk->dfailedFlag), sizeof(bool));
	hipMemset(mbk->dfailedFlag, 0, sizeof(bool));


	// Calling initPaging
	initPaging(mbk->availableGPUMemory, mbk);

	hashtableInit(NUM_BUCKETS, mbk, mbk->groupSize);
	
	
	printf("@INFO: transferring config structs to GPU memory\n");

	hipMalloc((void**) &(mbk->dstates), mbk->numRecords * sizeof(char));
	hipMemset(mbk->dstates, 0, mbk->numRecords * sizeof(char));


	mbk->myNumbers = (int*) malloc(2 * numThreads * sizeof(int));
	hipMalloc((void**) &(mbk->dmyNumbers), 2 * numThreads * sizeof(int));
	hipMemset((mbk->dmyNumbers), 0, 2 * numThreads * sizeof(int));

	size_t total, free;
	hipMemGetInfo(&free, &total);
	printf("total memory: %luMB, free: %luMB\n", total / (1 << 20), free / (1 << 20));


	printf("@INFO: number of page: %d\n", (int)(mbk->availableGPUMemory / PAGE_SIZE));
	printf("@INFO: number of hash groups: %d\n", mbk->numGroups);

	return mbk;
}

bool checkAndResetPass(multipassConfig_t* mbk, multipassConfig_t* dmbk)
{
	hipError_t errR = hipGetLastError();
	printf("#######Error at the beginning of checkAndReset: %s\n", hipGetErrorString(errR));

	hipMemcpy(mbk, dmbk, sizeof(multipassConfig_t), hipMemcpyDeviceToHost);
	bool failedFlag = false;
	int* hostCompleteFlag = mbk->hostCompleteFlag;
	int* gpuFlags = mbk->gpuFlags;
	bool* dfailedFlag = mbk->dfailedFlag;
	int* dmyNumbers = mbk->dmyNumbers;
	int* myNumbers = mbk->myNumbers;
	int flagSize = mbk->flagSize;
	void* hhashTableBaseAddr = mbk->hhashTableBaseAddr;
	largeInt hhashTableBufferSize = mbk->hhashTableBufferSize;
	int numThreads = mbk->numThreads;
	char* epochSuccessStatus = mbk->epochSuccessStatus;
	char* depochSuccessStatus = mbk->depochSuccessStatus;
	int epochNum = mbk->epochNum;

	hipMemcpy(epochSuccessStatus, depochSuccessStatus, epochNum * sizeof(char), hipMemcpyDeviceToHost);
	for(int i = 0; i < epochNum; i ++)
	{
		if(epochSuccessStatus[i] == UNTESTED)
			epochSuccessStatus[i] = SUCCEED;
		else if(epochSuccessStatus[i] == FAILED)
			epochSuccessStatus[i] = UNTESTED;
	}
	hipMemcpy(depochSuccessStatus, epochSuccessStatus, epochNum * sizeof(char), hipMemcpyHostToDevice);


	memset((void*) hostCompleteFlag, 0, flagSize);
	hipMemset(gpuFlags, 0, flagSize / 2);

	hipMemcpy(&failedFlag, dfailedFlag, sizeof(bool), hipMemcpyDeviceToHost);
	hipMemset(dfailedFlag, 0, sizeof(bool));

	hipMemcpy(mbk->hpages, mbk->pages, mbk->totalNumPages * sizeof(page_t), hipMemcpyDeviceToHost);

	
	hipMemcpy(mbk->hfreeListId, mbk->freeListId, mbk->totalNumPages * sizeof(int), hipMemcpyDeviceToHost);

	int freeListCounter = 0;
	int neededCounter = 0;
	int unneededCounter = 0;
	// Resetting the key page counter
	mbk->keyPageCounter = 0;
	for(int i = 0; i < mbk->totalNumPages; i ++)
	{
		if(mbk->hpages[i].needed == 0)
		{
			hipMemcpy((void*) ((largeInt) mbk->hpages[i].hashTableOffset + mbk->hpages[i].id * PAGE_SIZE), (void*) ((largeInt) mbk->dbuffer + mbk->hpages[i].id * PAGE_SIZE), PAGE_SIZE, hipMemcpyDeviceToHost);
			hipMemset((void*) ((largeInt) mbk->dbuffer + mbk->hpages[i].id * PAGE_SIZE), 0, PAGE_SIZE);

			mbk->hpages[i].hashTableOffset += mbk->totalNumPages * PAGE_SIZE;
			mbk->hpages[i].next = NULL;
			mbk->hpages[i].used = 0;

			mbk->hfreeListId[freeListCounter ++] = mbk->hpages[i].id;
			unneededCounter ++;
		}
		else
		{
			mbk->hpages[i].needed = 0;
			//printf("Page %d is needed..\n", i);
			neededCounter ++;
			//mbk->keyPageCounter ++;
		}
	}

	printf("@INFO: number of needed pages: %d, and number of unneededpages: %d (number of groups: %d)\n", neededCounter, unneededCounter, NUM_BUCKETS / mbk->groupSize);

	hipMemcpy(mbk->freeListId, mbk->hfreeListId, mbk->totalNumPages * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(mbk->pages, mbk->hpages, mbk->totalNumPages * sizeof(page_t), hipMemcpyHostToDevice);
	mbk->totalNumFreePages = freeListCounter;
	

	printf("totalnoPage * pagesize: %llu, hhashtbufferSize: %llu\n", (largeInt) mbk->totalNumPages * PAGE_SIZE, (largeInt) hhashTableBufferSize);
	mbk->hashTableOffset += mbk->totalNumPages * PAGE_SIZE;
	if((mbk->hashTableOffset + mbk->totalNumPages * PAGE_SIZE) > ((largeInt) hhashTableBaseAddr + hhashTableBufferSize) && failedFlag)
	{
		printf("Need more space on CPU memory for the hash table. Aborting...\n");
		exit(1);
	}


	mbk->initialPageAssignedCounter = 0;



	errR = hipGetLastError();
	printf("#######Error before setGroupPointer is: %s\n", hipGetErrorString(errR));

	printf("Before calling setGroupPointer, number of grids: %d\n", ((NUM_BUCKETS) + 1023) / 1024);
	setGroupsPointersDead<<<(((NUM_BUCKETS) + 1023) / 1024), 1024>>>(dmbk, NUM_BUCKETS);
	//setGroupsPointersDead<<<(((NUM_BUCKETS) + 256) / 255), 256>>>(mbk->groups, NUM_BUCKETS, GROUP_SIZE);
	hipDeviceSynchronize();

	errR = hipGetLastError();
	printf("#######Error after setGroupPointer is: %s\n", hipGetErrorString(errR));

	hipMemcpy(myNumbers, dmyNumbers, 2 * numThreads * sizeof(int), hipMemcpyDeviceToHost);
	hipMemset(dmyNumbers, 0, 2 * numThreads * sizeof(int));

	largeInt totalSuccess = 0;
	largeInt totalFailed = 0;
	for(int i = 0; i < numThreads; i ++)
	{
		totalSuccess += myNumbers[i * 2];
		totalFailed += myNumbers[i * 2 + 1];
	}

	printf("Total success: %lld\n", totalSuccess);
	printf("Total failure: %lld\n", totalFailed);

	hipMemcpy(dmbk, mbk, sizeof(multipassConfig_t), hipMemcpyHostToDevice);

	return failedFlag;
}

void* getKey(hashBucket_t* bucket)
{
	return (void*) ((largeInt) bucket + sizeof(hashBucket_t));
}

void* getValueHolder(hashBucket_t* bucket)
{
	int keySizeAligned = (bucket->keySize % ALIGNMET == 0)? bucket->keySize : bucket->keySize + (ALIGNMET - (bucket->keySize % ALIGNMET));
	return (void*) ((largeInt) bucket + sizeof(hashBucket_t) + keySizeAligned);
}

void* getValue(valueHolder_t* valueHolder)
{
	return (void*) ((largeInt) valueHolder + sizeof(valueHolder_t));
}

__device__ inline void setValue(valueHolder_t* valueHoder, void* value, int valueSize)
{
	for(int i = 0; i < (valueSize / ALIGNMET); i ++)
		*((largeInt*) ((largeInt) valueHoder + sizeof(valueHolder_t) + i * ALIGNMET)) = *((largeInt*) ((largeInt) value + i * ALIGNMET));
}
